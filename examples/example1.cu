#include "hip/hip_runtime.h"
#include "../src/fdm3d.cuh"
#include "../macros/errors.h"
#include <stdlib.h>
#include <stdio.h>





void cubeCreator(uint size, float *input) {
    for(int i = 0;i < size; ++i) {
        for(int j = 0; j < size; ++j){
            for(int k = 0; k < size; ++k){
                if(i != 0)
                    input[k + j*size + i*size*size] = 0.0f;
                else
                    input[k + j*size + i*size*size] = 100.0f;

            }
        }
    }
}

const int SIZE = 130;
const int STEPS = 5e4;
const int BLOCK_SIZE =16 ;
/////////////////////////////////////////
void dump(const float * const input, int size, int I) {
    if(input) {
        for(int i = 0;i < size; ++i) {
            for(int j = 0; j < size; ++j){
                for(int k = 0; k < size; ++k){
                   if( k == I ){
                   printf("%d %d %d %f \n",k,j,i,input[k+ j*size + i*size*size]);
                   if(j == size -1)
                    printf("\n");
                }
                }
            }
        }


    }
}


int main () {

    float* h_input = (float*)malloc(sizeof(float) * SIZE * SIZE *SIZE);

    float* d_input;
    float* d_output;


    cubeCreator(SIZE,h_input);

    CHECK_ERROR(hipMalloc((void**)(&d_input),sizeof(float) * SIZE*SIZE*SIZE));
    CHECK_ERROR(hipMalloc((void**)(&d_output),sizeof(float) * SIZE*SIZE*SIZE));

    CHECK_ERROR(hipMemcpy((void*)d_input,(void*)h_input,sizeof(float)*SIZE*SIZE*SIZE,hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy((void*)d_output,(void*)h_input,sizeof(float)*SIZE*SIZE*SIZE,hipMemcpyHostToDevice));

    for(int i = 0; i < STEPS; ++i) {
        fdm3d<<<dim3(8,8,1),dim3(BLOCK_SIZE,BLOCK_SIZE,1),3*(sizeof(float)*(BLOCK_SIZE+2)*(BLOCK_SIZE+2))>>>(d_input,d_output,SIZE,SIZE,make_float3(0.005f,0.005f,0.005f));
        CHECK_ERROR(hipDeviceSynchronize());

        float* d_temp = 0;
        d_temp = d_input;
        d_input = d_output;
        d_output = d_temp;

    }

    CHECK_ERROR(hipMemcpy((void*)h_input,(void*)d_input,sizeof(float)*SIZE*SIZE*SIZE,hipMemcpyDeviceToHost));

    dump(h_input,SIZE,SIZE/3);
    free(h_input);
    CHECK_ERROR(hipFree(d_input));
    CHECK_ERROR(hipFree(d_output));
    return 0;
}
